#include "hip/hip_runtime.h"
#include "CudaComputing.cuh"
#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include ""
#include "Lock.h"
#include "math.h"
//#include "windows.h"
#include "stdio.h"

//#include "Dian.h"
#include <afx.h>
#include "book.h"

//#define SIZE (100*1024*1024)
#define SIZE (Amount)//����൱�����������������
#define ELEMENTS (Amount)//�������pool��1000���������죬��������,�������ƥ���������������û������
#define HASH_ENTRIES 4000//���̫���˵����ٶ�̫�������ֻ��������Ͱ����������ԭ���Ǹ����������ϵ�

const int Amount = 2000;
const int length=24;
const int width=14;
const int height=20;//���Լ�һ������λ�ñ���
const int inEvery=2;
float singleBian=0.5;

//sph ����
const int JiaoDianNum=(length*inEvery+1)*(width*inEvery+1)*(height*inEvery+1);
const int NormalNum=((length-1)*inEvery+1)*((width-1)*inEvery+1)*((height-1)*inEvery+1);
const float Pi=3.1415926f;

//Voxel voxels[1500];

//Vec pointNormal[JiaoDianNum];
//float pointValue[JiaoDianNum];//�ǵ��valueֵ

Table myTable;//����Ϊȫ�ֱ��������������㷨�������Լ���ѹ�������ܶ�ѽ��������.����Ҳ�ò�������Ϊȫ�ֵı����ˣ�һ�ж������н����ˣ��ѵ�����������
              //��ȫ�ֱ���������һ���ô�������ֻ����һ�Σ�����ÿ��timer������һ��
void initialSPH(){
 
}


//����ͬʱ�����������豸�ĺ����������汾
__device__ __host__ size_t hash(unsigned int value,size_t count){
    return value%count;
}

__device__ __host__ int pothash(Vec myPot){//����
    //return value%count;
	int x=(int)floor(myPot.x*100);
	int y=(int)floor(myPot.y*100);
	int z=(int)floor(myPot.z*100);
	return x+20*y+200*z;
}

void initialize_table(Table &table,int entries,int elements){
	table.count=entries;
	hipMalloc((Entry**)&table.entries,entries*sizeof(Entry*));//����һ��Ͱ�����һ���ڵ��
	hipMemset(table.entries,0,entries*sizeof(Entry*));
	hipMalloc((Entry**)&table.pool,elements*sizeof(Entry));//////////////////////////////////
}

void free_table(Table &table){
	hipFree(table.pool);
	hipFree(table.entries);
}

__global__ void fill_entries(Table table,Table hostTable)
{
	int tid = threadIdx.x+blockIdx.x*blockDim.x;
	int stride=blockDim.x*gridDim.x;
	while(tid<table.count){
		if(hostTable.entries[tid]!=NULL)
			hostTable.entries[tid]=(Entry*)((size_t)hostTable.entries[tid]-(size_t)table.pool+(size_t)hostTable.pool);
		tid+=stride;
	}
}

//�������裬ǰ������Լ򵥣���һ����Ϊɢ�б����ݷ��������ڴ棬�ڶ������ܹ�cuadMemcpy()��GPU�ϵ����ݸ��Ƶ�����ڴ档
void copy_table_to_host(const Table &table,Table &hostTable){
	hostTable.count=table.count;
	hostTable.entries=(Entry**)calloc(table.count,sizeof(Entry*));
	hostTable.pool=(Entry*)malloc(ELEMENTS*sizeof(Entry));////////////////////////////////////////////////

	hipMemcpy(hostTable.entries,table.entries,table.count*sizeof(Entry*),hipMemcpyDeviceToHost);
	hipMemcpy(hostTable.pool,table.pool,ELEMENTS*sizeof(Entry),hipMemcpyDeviceToHost);//////////////////////////////

	/*�����������ܼ򵥵ؽ���Щָ�븴�Ƶ���������Щָ��ָ��ĵ�ַ��GPU�ϣ������������ϲ�����Чָ�롣����ָ������
	ƫ����Ȼ����Ч�ġ���Ч������ָ��Ϊ��x-table.pool)+hostTable.pool,��Ϊ������*/

	//�������������Ӱ���ٶȵĹؼ�
	for(int i=0;i<table.count;i++){
		if(hostTable.entries[i]!=NULL)
			hostTable.entries[i]=(Entry*)((size_t)hostTable.entries[i]-(size_t)table.pool+(size_t)hostTable.pool);
	}

	/*fill_entries<<<60,256>>>(table,hostTable);*/
	//�ǲ��ǿ��Կ��ǽ���Χ����
	for(int i=0;i<ELEMENTS;i++){
		if(hostTable.pool[i].next!=NULL)/////////////////////////////////////////
			hostTable.pool[i].next=(Entry*)((size_t)hostTable.pool[i].next-(size_t)table.pool+(size_t)hostTable.pool);
	}
}

void verify_table(const Table &dev_table){
	Table myTable;
	copy_table_to_host(dev_table,myTable);

	int count=0;
	for(size_t i=0;i<myTable.count;i++){
        
		Entry *current = myTable.entries[i];
		while(current!=NULL){
			++count;
			//if(pothash(current->pot)!=i)TRACE("%d hashed to %1d, but was located at %1d\n",current->key, pothash(current->pot),i);//Ϊʲô�������������أ���������
			
			//unsigned int temp=current->key;
			TRACE("%d hashed to %1d, and was located at %1d\n",current->key, pothash(current->pot),i);
			//TRACE("i is %d",i);
			//TRACE("key is %d",temp);

			current = current->next;
	    }
	}
			TRACE("Count is %d",count);

	/*if(count!=ELEMENTS)
		printf("%d elements found in hash table.Should be %1d\n",count,ELEMENTS);
	else
		printf("all %d elements found in has table.\n",count);*/
	free(myTable.pool);
	free(myTable.entries);
}

//����һ�������飬һ��ֵ���飬ɢ�б����Լ�һ�������顣���������������������飬��Ҫ���������Ի�
__global__ void add_to_table(Vec *particles,Table table,Lock *lock){
	int tid = threadIdx.x+blockIdx.x*blockDim.x;
	int stride=blockDim.x*gridDim.x;

	//�������飬��������ÿ�����ܹ�ɢ�к�������ϣ�����������������ĸ�Ͱ�������Ŀ��Ͱ���̻߳��������Ͱ����Ӻ󣬽������Ͱ

	while(tid<ELEMENTS){
		//unsigned int key = keys[tid];
		Vec tempDian = particles[tid];
		int hashValue = pothash(tempDian);//�ⲿһ��������һЩ�𣿣���
        //int hashValue = tempDian.value;
		//TRACE("value is %d",hashValue);

		//�߳�����һ������32���̵߳ļ��ϣ�������Щ�߳��Բ���һ�µķ�ʽִ�У�ÿ�����߳�����ֻ��һ���߳̿��Ի�ȡ���������ֹ32���߳���ͬʱ����
		for(int i=0;i<32;i++){
			if((tid%32)==i){
				//���������Ӷ��ڷ�Χ�ڻ
                Entry *location=&(table.pool[tid]);///////////////////////////////////////////////////////
				location->key=tid;
				location->pot=tempDian;

				lock[hashValue].lock();
				location->next = table.entries[hashValue];
				table.entries[hashValue] = location;
				lock[hashValue].unlock();
			}
		}
		tid+=stride;
	}
}

int* big_random_block( int size ) {
    int *data = (int*)malloc( size * sizeof(int) );
    //HANDLE_NULL( data );
    for (int i=0; i<size; i++)
        data[i] = rand();

    return data;
}
//vector3D *MyWall,point *r,float *p,float *pr,int number,float m,float po,float h)
__global__ void compute_D_P(Vec *particles,Table table,float *pr,float *p){
	int tid=threadIdx.x+blockIdx.x*blockDim.x;
    //SPH�еĹ̶�����
	float h=0.012;
	float m=0.001;
	float po=1000;
    size_t index;
	Vec myPot;
	Vec tempPot;

	while(tid<Amount){
		//�����ۼӣ��м�������ݴ�
        float sum=0;
		float tempR;
		float temp;
		
		myPot=particles[tid];
		
		//����ȷ�������ڿ�
		int a=(int)floor(myPot.x*100);
	    int b=(int)floor(myPot.y*100);
	    int c=(int)floor(myPot.z*100);

		for(int i=a-1;i<=a+1;i++){
			for(int j=b-1;j<=b+1;j++){
				for(int k=c-1;k<=c+1;k++){
					if(i>=0&&i<=19&&j>=0&&j<=9&&k>=0&&k<=19){//���仯
					   index=k*200+j*20+i;//���仯
					   Entry *current = table.entries[index];
		               while(current!=NULL){
			             if(current->key!=tid){
							 tempPot=current->pot;
						     tempR=sqrt((tempPot.x-myPot.x)*(tempPot.x-myPot.x)+(tempPot.y-myPot.y)*(tempPot.y-myPot.y)+(tempPot.z-myPot.z)*(tempPot.z-myPot.z));
							 if(tempR<=h){
                               temp=h*h-tempR*tempR;
			                   sum+=m*pow(temp,3);			
							 }
						 }
			             current = current->next;
	                   }
					}
				}
			}
		}
		//pr[tid]=1*315/(64*3.1415926*pow(h,9));
		pr[tid]=sum*315/(64*3.1415926*pow(h,9));

		p[tid]=1*(pr[tid]-po);//���������Ƚ�

        tid+=blockDim.x*gridDim.x;
	}
}

__global__ void compute_A(Vec *particles,Vec *velocity,Vec *acceleration,Table table,float *pr,float *p){
	int tid=threadIdx.x+blockIdx.x*blockDim.x;
    //SPH�еĹ̶�����
	float h=0.012;
	float m=0.001;
	float ur=3;
	float g=9.8;
	float Pi=3.1415926;
	size_t index;

	while(tid<Amount){
		//�����ۼӣ��м�������ݴ�
		float sumX=0;
		float sumY=0;
		float sumZ=0;

       // float sum=0;
		float tempR;
		//float temp;
		
      
		Vec myPot=particles[tid];
		Vec tempPot;
		int tempKey;
		//����ȷ�������ڿ�
		int a=(int)floor(myPot.x*100);
	    int b=(int)floor(myPot.y*100);
	    int c=(int)floor(myPot.z*100);

		for(int i=a-1;i<=a+1;i++){
			for(int j=b-1;j<=b+1;j++){
				for(int k=c-1;k<=c+1;k++){
					if(i>=0&&i<=19&&j>=0&&j<=9&&k>=0&&k<=19){//���ݷ�Χ�仯�仯
					   index=k*200+j*20+i;
					   Entry *current = table.entries[index];
		               while(current!=NULL){
			             if(current->key!=tid){
							 tempPot=current->pot;
							 tempKey=current->key;
						     tempR=sqrt((tempPot.x-myPot.x)*(tempPot.x-myPot.x)+(tempPot.y-myPot.y)*(tempPot.y-myPot.y)+(tempPot.z-myPot.z)*(tempPot.z-myPot.z));
							 if(tempR<=h){
                               sumX+=(45*m/(Pi*pow(h,6)))*((p[tid]+p[tempKey])*pow((h-tempR),2)*(myPot.x-tempPot.x))/(2*pr[tid]*pr[tempKey])/tempR;
				               sumY+=(45*m/(Pi*pow(h,6)))*((p[tid]+p[tempKey])*pow((h-tempR),2)*(myPot.y-tempPot.y))/(2*pr[tid]*pr[tempKey])/tempR;
				               sumZ+=(45*m/(Pi*pow(h,6)))*((p[tid]+p[tempKey])*pow((h-tempR),2)*(myPot.z-tempPot.z))/(2*pr[tid]*pr[tempKey])/tempR;

				               sumX+=(45*m*ur/(Pi*pow(h,6)))*(velocity[tempKey].x-velocity[tid].x)*(h-tempR)/(pr[tid]*pr[tempKey]);
				               sumY+=(45*m*ur/(Pi*pow(h,6)))*(velocity[tempKey].y-velocity[tid].y)*(h-tempR)/(pr[tid]*pr[tempKey]);
				               sumZ+=(45*m*ur/(Pi*pow(h,6)))*(velocity[tempKey].z-velocity[tid].z)*(h-tempR)/(pr[tid]*pr[tempKey]);
				 			 }
						 }
			             current = current->next;
	                   }
					}
				}
			}
		}
		//sumX=0;
		//sumY=0;
		//sumZ=0;

		acceleration[tid].x=sumX;
 
 		acceleration[tid].y=sumY;
 
 		acceleration[tid].z=sumZ-g;

        tid+=blockDim.x*gridDim.x;
	}
}

__global__ void compute_V(Vec *velocity,Vec *acceleration){//û����
	int tid=threadIdx.x+blockIdx.x*blockDim.x;
    while(tid<Amount){
        velocity[tid].x=velocity[tid].x+acceleration[tid].x*1/1000;
        velocity[tid].y=velocity[tid].y+acceleration[tid].y*1/1000;
        velocity[tid].z=velocity[tid].z+acceleration[tid].z*1/1000;
        tid+=blockDim.x*gridDim.x;
	}
}

__global__ void compute_P(Vec *particles,Vec *velocity,Vec *acceleration){//û����
	int tid=threadIdx.x+blockIdx.x*blockDim.x;
    while(tid<Amount){
	    
		particles[tid].x=particles[tid].x+velocity[tid].x*1/1000;
		particles[tid].y=particles[tid].y+velocity[tid].y*1/1000;
		particles[tid].z=particles[tid].z+velocity[tid].z*1/1000;


		//���α߽�,�������ò���

		if(particles[tid].x<=0.00){
			particles[tid].x=0.00;
			velocity[tid].x=-velocity[tid].x/2;
			//acceleration[tid].x=0;
		}
		if(particles[tid].x>=0.199){
			particles[tid].x=0.199;
		    velocity[tid].x=-velocity[tid].x/2;
			//acceleration[tid].x=0;
		}
		if(particles[tid].y<=0.00){
			particles[tid].y=0.00;
			velocity[tid].y=-velocity[tid].y/2;
			//acceleration[tid].y=0;
		}
		if(particles[tid].y>=0.099){
			particles[tid].y=0.099;
			velocity[tid].y=-velocity[tid].y/2;
			//acceleration[tid].y=0;
		}

		if(particles[tid].z<=0.00){
			particles[tid].z=0.00;
			velocity[tid].z=-velocity[tid].z/3;//֮ǰͣ��������Ϊ���˽���
			acceleration[tid].z=-acceleration[tid].z;
		}
		if(particles[tid].z>=0.199){
			particles[tid].z=0.199;
			velocity[tid].z=-velocity[tid].z/2;//֮ǰͣ��������Ϊ���˽���
			//acceleration[tid].z=0;
		}
		tid+=blockDim.x*gridDim.x;
	}
}

//����Ĺؼ��ǣ������ĵ��ǳ���100�Ժ��ֵ�����������õ����ٳ˻�һ�ٵ�ֵ

__global__ void computeJiaoDianValue(Vec *particles,float *pointValue){//���Ӧ�ü���table
	int tid=threadIdx.x+blockIdx.x*blockDim.x;
	//Point point;
	int a;
	int b;
	int c;

    float px;
	float py;
	float pz;

    float value;
    float radius;
    float R=1.3;

	int divZ=(length*inEvery+1)*(width*inEvery+1);
    int divY=(length*inEvery+1);
	float singleBian=0.5;

	if(tid<JiaoDianNum){
	   value=0;

	   /*���Ըĳɳ����*/
	   c=floor((float)tid/divZ);
	   b=floor(((float)tid-divZ*c)/divY);
	   a=tid-b*divY-c*divZ;

	   px=(float)a*singleBian;//�������õķ�ΧҪ��
	   py=(float)b*singleBian;
	   pz=(float)c*singleBian;

	   for(int j=0;j<Amount;j++){//̫�����˰�����Ҫ��
	     radius=sqrt((px-particles[j].x)*(px-particles[j].x)+(py-particles[j].y)*(py-particles[j].y)+(pz-particles[j].z)*(pz-particles[j].z));
	     if(radius<=R){
	         value+=1000*(1-(radius/R)*(radius/R))*(1-(radius/R)*(radius/R));
	     }
	   }

	   value-=500;
       pointValue[tid]=value;

	   //tid+=blockIdx.x*blockDim.x;
    }   
}

//Ҫ��Ҫ��Ҫ�ģ�һ��r,һ��point Value
void computeAllValue(Vec *particles,float *pointValue){
  // Voxel *dev_vox;
   Vec *dev_particles;
   float *dev_value;

   //GPU����ռ�
   //hipMalloc((Voxel**)&dev_vox,1500*sizeof(Voxel));
   hipMalloc((void**)&dev_particles,Amount*sizeof(Vec));
   hipMalloc((void**)&dev_value,JiaoDianNum*sizeof(float));

   //�������ݵ�GPU
   
  // hipMemcpy(dev_vox,voxels,1500*sizeof(Voxel),hipMemcpyHostToDevice);
   hipMemcpy(dev_particles,particles,Amount*sizeof(Vec),hipMemcpyHostToDevice);
   hipMemcpy(dev_value,pointValue,JiaoDianNum*sizeof(float),hipMemcpyHostToDevice);


   //��ʼ������ 
   
   computeJiaoDianValue<<<1000,512>>>(dev_particles,dev_value);
   
   //GPU���ƻ�CPU
   //hipMemcpy(voxels,dev_vox,1500*sizeof(Voxel),hipMemcpyDeviceToHost);
   hipMemcpy(particles,dev_particles,Amount*sizeof(Vec),hipMemcpyDeviceToHost);
   hipMemcpy(pointValue,dev_value,JiaoDianNum*sizeof(float),hipMemcpyDeviceToHost);

   //�ͷ��ڴ�
   //hipFree(dev_vox);
   hipFree(dev_particles);
   hipFree(dev_value);
}

__global__ void computeSingleNormal(Vec *pointNormal,float *pointValue){
//�ؼ�Ҫ���Ǻñ߽������
   int i=threadIdx.x+blockIdx.x*blockDim.x;//ע��i
   int a;
   int b;
   int c;
   int divY=(length*inEvery+1);
   int divZ=(length*inEvery+1)*(width*inEvery+1);
   float singleBian=0.5;
   if(i<JiaoDianNum){
      c=floor((float)i/(float)divZ);
      b=floor(((float)i-(float)divZ*c)/(float)divY);
      a=i-b*divY-c*divZ;
	  if(a>0&&a<length*inEvery+1){
	    pointNormal[i].x=(pointValue[i+1]-pointValue[i-1])/(2*singleBian);
	  }
	  if(a==0&&a==length*inEvery+1){
	    pointNormal[i].x=0;
	  }
	  if(b>0&&b<width*inEvery+1){
	    pointNormal[i].y=(pointValue[i+divY]-pointValue[i-divY])/(2*singleBian);
	  }
	  if(b==0&&b==width*inEvery+1){
	    pointNormal[i].y=0;
	  }
	  if(c>0&&c<height*inEvery+1){
	    pointNormal[i].z=(pointValue[i+divZ]-pointValue[i-divZ])/(2*singleBian);
	  }
	  if(c==0&&c==height*inEvery+1){
	    pointNormal[i].z=0;
	  }
    //i+=blockDim.x*gridDim.x;
   }
}

void computeAllNormal(Vec *pointNormal,float *pointValue){
   Vec *dev_normal;
   float *dev_value;

   //GPU����ռ�
   hipMalloc((Vec**)&dev_normal,JiaoDianNum*sizeof(Vec));
   hipMalloc((float**)&dev_value,JiaoDianNum*sizeof(float));

   //�������ݵ�GPU
   
  // hipMemcpy(dev_vox,voxels,1500*sizeof(Voxel),hipMemcpyHostToDevice);
   hipMemcpy(dev_normal,pointNormal,JiaoDianNum*sizeof(Vec),hipMemcpyHostToDevice);
   hipMemcpy(dev_value,pointValue,JiaoDianNum*sizeof(float),hipMemcpyHostToDevice);


   //��ʼ������ 
   

   //29435
   //computSingleV<<<150,128>>>(dev_vox,dev_r,dev_value);
  // computeJiaoDianValue<<<1000,1024>>>(dev_normal,dev_value);
   computeSingleNormal<<<1000,512>>>(dev_normal,dev_value);


   //GPU���ƻ�CPU
   //hipMemcpy(voxels,dev_vox,1500*sizeof(Voxel),hipMemcpyDeviceToHost);
   hipMemcpy(pointNormal,dev_normal,JiaoDianNum*sizeof(Vec),hipMemcpyDeviceToHost);
   hipMemcpy(pointValue,dev_value,JiaoDianNum*sizeof(float),hipMemcpyDeviceToHost);

   //�ͷ��ڴ�
   //hipFree(dev_vox);
   hipFree(dev_normal);
   hipFree(dev_value);

}

Lock lock[HASH_ENTRIES];

int get_table(Vec *particles,Vec *velocities,Vec *accelerations,float *densities,float *pressures){
	//unsigned int buffer[10]={

	//hipEvent_t start,stop;

	//hipEventCreate(&start);
	//hipEventCreate(&stop);
	//hipEventRecord(start,0);


	//unsigned int *buffer = (unsigned int*)big_random_block(SIZE);//���������
	//unsigned int *dev_keys;
	//void **dev_values;
	//hipMalloc((void**)&dev_keys,SIZE*sizeof(unsigned int));
	//hipMalloc((void**)&dev_values,SIZE);
	//hipMemcpy(dev_keys,buffer,SIZE*sizeof(unsigned int),hipMemcpyHostToDevice);//����
    Table table;//���table����GPU�����ռ��
	initialize_table(table,HASH_ENTRIES,ELEMENTS);

	Vec *dev_particles;
	hipMalloc((Vec**)&dev_particles,Amount*sizeof(Vec));//����ɵ����,֮ǰ����Ŀռ�̫С��
	hipMemcpy(dev_particles,particles,Amount*sizeof(Vec),hipMemcpyHostToDevice);//����

	

	//Ϊɢ�б�Ͱ��������ÿһ��Ͱ����һ����������ֻ��һ�����������ή�����ܣ���Ҫ����
	Lock *dev_lock;
	hipMalloc((Lock**)&dev_lock,HASH_ENTRIES*sizeof(Lock));
	hipMemcpy(dev_lock,lock,HASH_ENTRIES*sizeof(Lock),hipMemcpyHostToDevice);
	
	float *dev_density;
	hipMalloc((float**)&dev_density,Amount*sizeof(float));
	hipMemcpy(dev_density,densities,Amount*sizeof(float),hipMemcpyHostToDevice);

    float *dev_press;
	hipMalloc((float**)&dev_press,Amount*sizeof(float));
	hipMemcpy(dev_press,pressures,Amount*sizeof(float),hipMemcpyHostToDevice);

    Vec *dev_vel;
	hipMalloc((Vec**)&dev_vel,Amount*sizeof(Vec));
	hipMemcpy(dev_vel,velocities,Amount*sizeof(Vec),hipMemcpyHostToDevice);

	Vec *dev_accel;
	hipMalloc((Vec**)&dev_accel,Amount*sizeof(Vec));
	hipMemcpy(dev_accel,accelerations,Amount*sizeof(Vec),hipMemcpyHostToDevice);
	//����ӵ�ɢ�б�
	add_to_table<<<60,256>>>(dev_particles,table,dev_lock);

	//hipEventRecord(stop,0);
	//hipEventSynchronize(stop);
//	float elapsedTime;
//	hipEventElapsedTime(&elapsedTime,start,stop);/////////////////////////////////////////

	//printf("Time to hash:%3.1f ms\n",elapsedTime);

	//I***************************************
	//verify_table(table);//��Ҫ���Ͱ�����Թ���������ʹ�ã�����������ж�λ�ú�����ֵû�иı�

	//hipEventDestroy(start);
	//hipEventDestroy(stop);
	
	compute_D_P<<<60,256>>>(dev_particles,table,dev_density,dev_press);

	/*float den[Amount];

	hipMemcpy(den,dev_density,Amount*sizeof(float),hipMemcpyDeviceToHost);

	for(int i=0;i<Amount;i++){
	    den[i];
	}*/

	compute_A<<<60,256>>>(dev_particles,dev_vel,dev_accel,table,dev_density,dev_press);

	compute_V<<<60,256>>>(dev_vel,dev_accel);

	compute_P<<<60,256>>>(dev_particles,dev_vel,dev_accel);
	//����������GPU�д��ݣ���󴫳�λ�õ����ݾ���
	//��һ���ļ���value�Ⱦ�Ҳ���Խ������ڽӱ������Լ������ͣ�������ƺܺ�

	hipMemcpy(accelerations,dev_accel,Amount*sizeof(Vec),hipMemcpyDeviceToHost);

	hipMemcpy(velocities,dev_vel,Amount*sizeof(Vec),hipMemcpyDeviceToHost);

	hipMemcpy(particles,dev_particles,Amount*sizeof(Vec),hipMemcpyDeviceToHost);

	

	free_table(table);

	hipFree(dev_lock);
	hipFree(dev_particles);
	hipFree(dev_density);
	hipFree(dev_press);
	hipFree(dev_vel);
	hipFree(dev_accel);

	//֮ǰ������table lock particles�������ռ����Ҫ�ͷŵ�
	//hipFree(dev_keys);
	//hipFree(dev_values);
	//free(buffer);
	return 0;
}

__global__ void dot(float* a,float* b,float* c){
  __shared__ float cache[threadsPerBlock];//����������shared��
  int tid=threadIdx.x+blockIdx.x*blockDim.x;
  int cacheIndex=threadIdx.x;

  float temp=0;
  while(tid<N){
	  temp +=a[tid]*b[tid];
	  tid +=blockDim.x*gridDim.x;
  }

  cache[cacheIndex]=temp;

  __syncthreads();

  int i=blockDim.x/2;

  while(i!=0){//��Լ����
     if(cacheIndex<i){
		 cache[cacheIndex]+=cache[cacheIndex+i];
	 }
	 __syncthreads();
	 i/=2;
  }
  if(cacheIndex==0){//���鵽0����
	  c[blockIdx.x]=cache[0];
  }
}

void MultDot(float *a,float *b,float *partial_c){
	float *dev_a,*dev_b,*dev_partial_c;

	hipMalloc((float**)&dev_a,N*sizeof(float));
	hipMalloc((float**)&dev_b,N*sizeof(float));
	hipMalloc((float**)&dev_partial_c,blocksPerGrid*sizeof(float));

	hipMemcpy(dev_a,a,N*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(dev_b,b,N*sizeof(float),hipMemcpyHostToDevice);
	//hipMemcpy(
	
	dot<<<blocksPerGrid,threadsPerBlock>>>(dev_a,dev_b,dev_partial_c);

	hipMemcpy(partial_c,dev_partial_c,blocksPerGrid*sizeof(float),hipMemcpyDeviceToHost);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_partial_c);
}

/*
__global__ void getEveryNear(point *r,int number,float h){
  int i=threadIdx.x+blockIdx.x*blockDim.x;
  int j=threadIdx.y+blockIdx.y*blockDim.y;
   
 // float h=0.0055;//����ĿҪ��

   while(i<number){
	   r[i].index=0;
       int k=j;
	   while(k<number){
		   float tempR;
		   
		   tempR=sqrt((r[i].x-r[k].x)*(r[i].x-r[k].x)+(r[i].y-r[k].y)*(r[i].y-r[k].y)+(r[i].z-r[k].z)*(r[i].z-r[k].z));
		   
	    if(tempR<h){
	     int tempIndex=r[i].index;
		 r[i].points[tempIndex]=k;
		 tempIndex++;
		 r[i].index=tempIndex;
		 if(tempIndex>12){
		 r[i].index=12;
		 }
	   } 
	   k+=blockDim.y*gridDim.y;
	  }
	 i+=blockDim.x*gridDim.x;
   }    
	   

	  
}
void getNearPoint(point *r,int number,float h){
   
   point *dev_r;
  
   hipMalloc((point**)&dev_r,number*sizeof(point));
  
   
   hipMemcpy(dev_r,r,number*sizeof(point),hipMemcpyHostToDevice);
   //��ʼ������ 
   
   getEveryNear<<<100,128>>>(dev_r,number,h);
   //fillVoxels<<<200,10>>>();
   
   //GPU���ƻ�CPU
   hipMemcpy(r,dev_r,number*sizeof(point),hipMemcpyDeviceToHost);
   //�ͷ��ڴ�
   hipFree(dev_r);
}
*/

/*
__global__ void computeTestKernel(Grid* g,int* test){
	int i=threadIdx.x;
	Node* tempNode;
	int temp;
	if(i<100){
	  temp=test[i]%8;

	  //tempNode=(Node*)malloc(sizeof(Node));

	 

      tempNode->num=test[i];
	  tempNode->next=g[i].first;
	  g[i].first->next=tempNode;
	  //g[temp].num=test[i];
	  
	}
}


void computeTest(Grid* g,int n1,int* test,int n2)
{
	Grid* dev_g;
	int* dev_test;

	hipMalloc((Grid**)&dev_g,n1*sizeof(Grid));
	hipMalloc((int**)&dev_test,n2*sizeof(int));

	hipMemcpy(dev_g,g,n1*sizeof(Grid),hipMemcpyHostToDevice);
	hipMemcpy(dev_test,test,n2*sizeof(int),hipMemcpyHostToDevice);

	
	computeTestKernel<<<100,1>>>(g,test);


	hipMemcpy(g,dev_g,n1*sizeof(Grid),hipMemcpyDeviceToHost);
 
	hipFree(dev_g);
	hipFree(dev_test);
}
*/
                  